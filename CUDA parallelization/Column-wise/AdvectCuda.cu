#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include "AdvectCoeff.h"
#include "Advect3D.h"

#undef V
#undef Vh
#define V(u, i, j) (*((u) -> ix(i, j, 0)))
#define Vh(u, i, j) (*((u) -> ix_h(i, j, 0)))
#define VT(u, i, j, T) (*((u) -> ix(i, j, T)))
#define VhT(u, i, j, T) (*((u) -> ix_h(i, j, T)))
extern bool opt_tmr;
extern int gdim, bdim;

__device__ double median3cuda(double a, double b, double c){
    if ( a < b ) {
        if ( b < c ) return b;
        if ( a > c ) return a;
        else return c;
    }else { // b < a
        if ( a < c ) return a;
        else if ( b > c) return b;
        else return c;
    }
}

__global__ void boundary(HaloArray3D* u){
    int lx = u->l.x, ly = u->l.y, sx = u->s.x;
    int hx = u->halo.x, hy = u->halo.y;


    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;

    for(int j = j0; j < ly + hy; j+= dj){
        for(int ib = i0; ib < hx; ib += di) {
            V(u, ib, j) = V(u, lx + ib - u->B, j);
            V(u, lx + hx + ib, j) = V(u, hx + ib + u->B, j);
        }
    }

    for(int j = j0; j < hy; j += dj){
        for(int i = i0; i < sx; i += di){
            V(u, i, j) = V(u, i, ly + j - 1);
            V(u, i, ly + hy + j) = V(u, i, hy + j + 1);
        }
    }
}

__global__ void LW2DtmrKernel1(HaloArray3D* u){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    for(int j = j0; j < u->s.y; j += dj){
        for(int i = i0; i < u->s.x; i += di){
            VT(u, i, j, 1) = VT(u, i, j, 2) =  VT(u, i, j, 0);
        }
    }
}

__global__ void LW2DtmrKernel2(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy, int t){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    for(int j = j0; j < u->l.y; j += dj) {
        for (int i = i0; i < u->l.x; i += di) {
            V(uh, i, j) = Vy * (Vy - 1.0) * (Ux * (Ux - 1.0) * VhT(u, i + 1, j + 1, t) / 2
                                             + Ux * (Ux + 1.0) * VhT(u, i - 1, j + 1, t) / 2
                                             + (1.0 - Ux * Ux) * VhT(u, i, j + 1, t)) / 2
                          + Vy * (Vy + 1.0) * (Ux * (Ux - 1.0) * VhT(u, i + 1, j - 1, t) / 2
                                               + Ux * (Ux + 1.0) * VhT(u, i - 1, j - 1, t) / 2
                                               + (1.0 - Ux * Ux) * VhT(u, i, j - 1, t)) / 2
                          + (1.0 - Vy * Vy) * (Ux * (Ux - 1.0) * VhT(u, i + 1, j, t) / 2
                                               + Ux * (Ux + 1.0) * VhT(u, i - 1, j, t) / 2
                                               + (1.0 - Ux * Ux) * VhT(u, i, j, t));
        }
    }
}

__global__ void LW2DtmrKernel3(HaloArray3D* u, HaloArray3D* uh1, HaloArray3D* uh2, HaloArray3D* uh3){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    for(int j = j0; j < u->l.y; j += dj) {
        for (int i = i0; i < u->l.x; i += di) {
            VhT(u, i, j, 0) = median3cuda(Vh(uh1, i, j), Vh(uh2, i, j), Vh(uh3, i, j));
        }
    }
}

void Advect3D::updateLW2D_tmr_Cuda(HaloArray3D* u){
    HaloArray3D *uh[3];
    const dim3 gridSize(gdim, gdim, 1);
    const dim3 blockSize(bdim, bdim, 1);

    for(int t = 0; t < 3; t++){
        uh[t] = new HaloArray3D(Vec3D<int>(u->s.x - 1, u->s.y - 1, 1), Vec3D<int>(0), B);
    }
    double Ux = V.x * dt / delta.x, Vy = V.y * dt / delta.y;

    LW2DtmrKernel1<<<gridSize, blockSize>>>(u);
    hipDeviceSynchronize();

    HaloArray3D* d_uh1, *d_uh2, *d_uh3;
    double* uuh1, *uuh2, *uuh3;
    hipMallocManaged((void**) &d_uh1, sizeof(HaloArray3D));
    hipMallocManaged((void**) &d_uh2, sizeof(HaloArray3D));
    hipMallocManaged((void**) &d_uh3, sizeof(HaloArray3D));
    hipMallocManaged((void**) &uuh1, sizeof(double) * uh[0]->s.prod() * 3);
    hipMallocManaged((void**) &uuh2, sizeof(double) * uh[1]->s.prod() * 3);
    hipMallocManaged((void**) &uuh3, sizeof(double) * uh[2]->s.prod() * 3);
    d_uh1->u = uuh1;
    d_uh1->l = uh[0]->l;
    d_uh1->s = uh[0]->s;
    d_uh1->halo = uh[0]->halo;
    d_uh1->B = uh[0]->B;

    d_uh2->u = uuh2;
    d_uh2->l = uh[1]->l;
    d_uh2->s = uh[1]->s;
    d_uh2->halo = uh[1]->halo;
    d_uh2->B = uh[1]->B;

    d_uh3->u = uuh3;
    d_uh3->l = uh[2]->l;
    d_uh3->s = uh[2]->s;
    d_uh3->halo = uh[2]->halo;
    d_uh3->B = uh[2]->B;


    LW2DtmrKernel2<<<gridSize, blockSize>>>(u, d_uh1, Ux, Vy, 0);
    hipDeviceSynchronize();
    LW2DtmrKernel2<<<gridSize, blockSize>>>(u, d_uh2, Ux, Vy, 1);
    hipDeviceSynchronize();
    LW2DtmrKernel2<<<gridSize, blockSize>>>(u, d_uh3, Ux, Vy, 2);
    hipDeviceSynchronize();
    LW2DtmrKernel3<<<gridSize, blockSize>>>(u, d_uh1, d_uh2, d_uh3);
    hipDeviceSynchronize();

    hipFree(d_uh1);
    hipFree(d_uh2);
    hipFree(d_uh3);
    hipFree(uuh1);
    hipFree(uuh2);
    hipFree(uuh3);
}


__global__ void LWN2kernel1(HaloArray3D* u, HaloArray3D* uh, double Ux, double Uy, int optimized){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            double cim1, ci0, cip1;
            double cjm1, cj0, cjp1;
            N2Coeff(Ux, cim1, ci0, cip1);
            N2Coeff(Uy, cjm1, cj0, cjp1);

            Vh(uh, i, j) = cim1 * (cjm1 * Vh(u, i - 1, j - 1) + cj0 * Vh(u, i - 1, j) + cjp1 * Vh(u, i - 1, j + 1)) +
                           ci0 * (cjm1 * Vh(u, i, j - 1) + cj0 * Vh(u, i, j) + cjp1 * Vh(u, i, j + 1)) +
                           cip1 * (cjm1 * Vh(u, i + 1, j - 1) + cj0 * Vh(u, i + 1, j) + cjp1 * Vh(u, i + 1, j + 1));
        }
    }
}

__global__ void LWN2kernel2(HaloArray3D* u, HaloArray3D* uh, int optimized){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            Vh(u, i, j) = Vh(uh, i, j);
        }
    }
}

void Advect3D::updateLWN2Cuda(HaloArray3D *u) {

    double Ux = V.x * dt / delta.x, Uy = V.y * dt / delta.y;

    HaloArray3D *uh = new HaloArray3D(u->l, Vec3D<int>(0), 1);
    int sizeuh = uh->s.prod();
    HaloArray3D* d_uh;
    double* uuh;
    hipMallocManaged((void**) &d_uh, sizeof(HaloArray3D));
    hipMallocManaged((void**) &uuh, sizeof(double) * sizeuh);
    d_uh->u = uuh;
    d_uh->l = uh->l;
    d_uh->s = uh->s;
    d_uh->halo = uh->halo;
    d_uh->B = uh->B;

    const dim3 gridSize(gdim, gdim, 1);
    const dim3 blockSize(bdim, bdim, 1);

    LWN2kernel1<<<gridSize, blockSize>>>(u, d_uh, Ux, Uy, 1);
    hipDeviceSynchronize();
    LWN2kernel2<<<gridSize, blockSize>>>(u, d_uh, 1);
    hipDeviceSynchronize();
    hipFree(d_uh);
    hipFree(uuh);
} //updateLWN2()

__global__ void LWN4kernel1(HaloArray3D* u, HaloArray3D* uh, double Ux, double Uy){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            double cim2, cim1, ci0, cip1, cip2;
            double cjm2, cjm1, cj0, cjp1, cjp2;
            N4Coeff(Ux, cim2, cim1, ci0, cip1, cip2);
            N4Coeff(Uy, cjm2, cjm1, cj0, cjp1, cjp2);
            Vh(uh, i, j) =
                    cim2 * (cjm2 * Vh(u, i - 2, j - 2) + cjm1 * Vh(u, i - 2, j - 1) + cj0 * Vh(u, i - 2, j) +
                            cjp1 * Vh(u, i - 2, j + 1) + cjp2 * Vh(u, i - 2, j + 2)) +
                    cim1 * (cjm2 * Vh(u, i - 1, j - 2) + cjm1 * Vh(u, i - 1, j - 1) + cj0 * Vh(u, i - 1, j) +
                            cjp1 * Vh(u, i - 1, j + 1) + cjp2 * Vh(u, i - 1, j + 2)) +
                    ci0 * (cjm2 * Vh(u, i, j - 2) + cjm1 * Vh(u, i, j - 1) + cj0 * Vh(u, i, j) +
                           cjp1 * Vh(u, i, j + 1) + cjp2 * Vh(u, i, j + 2)) +
                    cip1 * (cjm2 * Vh(u, i + 1, j - 2) + cjm1 * Vh(u, i + 1, j - 1) + cj0 * Vh(u, i + 1, j) +
                            cjp1 * Vh(u, i + 1, j + 1) + cjp2 * Vh(u, i + 1, j + 2)) +
                    cip2 * (cjm2 * Vh(u, i + 2, j - 2) + cjm1 * Vh(u, i + 2, j - 1) + cj0 * Vh(u, i + 2, j) +
                            cjp1 * Vh(u, i + 2, j + 1) + cjp2 * Vh(u, i + 2, j + 2));
        }
    }
}

__global__ void LWN4kernel2(HaloArray3D* u, HaloArray3D* uh){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            Vh(u, i, j) = Vh(uh, i, j);
        }
    }
}

void Advect3D::updateLWN4Cuda(HaloArray3D *u){
    const dim3 gridSize(gdim, gdim, 1);
    const dim3 blockSize(bdim, bdim, 1);
    double Ux = V.x * dt / delta.x, Uy = V.y * dt / delta.y;
    HaloArray3D *uh = new HaloArray3D(u->l, Vec3D<int>(0), 1);
    int sizeuh = uh->s.prod();
    HaloArray3D* d_uh;
    double* uuh;
    hipMallocManaged((void**) &d_uh, sizeof(HaloArray3D));
    hipMallocManaged((void**) &uuh, sizeof(double) * sizeuh);
    d_uh->u = uuh;
    d_uh->l = uh->l;
    d_uh->s = uh->s;
    d_uh->halo = uh->halo;
    d_uh->B = uh->B;

    LWN4kernel1<<<gridSize, blockSize>>>(u, d_uh, Ux, Uy);
    hipDeviceSynchronize();
    LWN4kernel2<<<gridSize, blockSize>>>(u, d_uh);
    hipDeviceSynchronize();
    hipFree(d_uh);
    hipFree(uuh);
}

__global__ void NN2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy - 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 1) / 2
                                             + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 1) / 2
                                             + (1.0 - Ux * Ux) * Vh(u, i, j + 1)) / 2
                          + Vy * (Vy + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 1) / 2
                                               + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 1) / 2
                                               + (1.0 - Ux * Ux) * Vh(u, i, j - 1)) / 2
                          + (1.0 - Vy * Vy) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j) / 2
                                               + Ux * (Ux + 1.0) * Vh(u, i - 1, j) / 2
                                               + (1.0 - Ux * Ux) * Vh(u, i, j));
        }
    }
}

__global__ void NW2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy / 2 - 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 2) / 2
                                                 + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 2) / 2
                                                 + (1.0 - Ux * Ux) * Vh(u, i, j + 2)) / 4
                          + Vy * (Vy / 2 + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 2) / 2
                                                   + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 2) / 2
                                                   + (1.0 - Ux * Ux) * Vh(u, i, j - 2)) / 4
                          + (-Vy * Vy / 4 + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j) / 2
                                                    + Ux * (Ux + 1.0) * Vh(u, i - 1, j) / 2
                                                    + (1.0 - Ux * Ux) * Vh(u, i, j));
        }
    }
}

__global__ void NF2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = (-Vy * Vy / 16 + 9.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 1) / 2
                                                        + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 1) / 2
                                                        + (1.0 - Ux * Ux) * Vh(u, i, j - 1))
                          + (-Vy * Vy / 16 + 9.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 1) / 2
                                                          + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 1) / 2
                                                          + (1.0 - Ux * Ux) * Vh(u, i, j + 1))
                          + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 3) / 2
                                                                  + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 3) / 2
                                                                  + (1.0 - Ux * Ux) * Vh(u, i, j + 3))
                          + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 3) / 2
                                                                  + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 3) / 2
                                                                  + (1.0 - Ux * Ux) * Vh(u, i, j - 3));
        }
    }
}

__global__ void WN2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy - 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 1) / 4
                                             + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 1) / 4
                                             + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 1)) / 2
                          + Vy * (Vy + 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 1) / 4
                                               + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 1) / 4
                                               + (-Ux * Ux / 4 + 1.0) * Vh(u, i, j - 1)) / 2
                          + (1.0 - Vy * Vy) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j) / 4
                                               + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j) / 4
                                               + (1.0 - Ux * Ux / 4) * Vh(u, i, j));
        }
    }
}

__global__ void WW2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy / 2 - 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 2) / 4
                                                 + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 2) / 4
                                                 + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 2)) / 4
                          + Vy * (Vy / 2 + 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 2) / 4
                                                   + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 2) / 4
                                                   + (1.0 - Ux * Ux / 4) * Vh(u, i, j - 2)) / 4
                          + (1.0 - Vy * Vy / 4) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j) / 4
                                                   + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j) / 4
                                                   + (1.0 - Ux * Ux / 4) * Vh(u, i, j));
        }
    }
}

__global__ void WF2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = (9.0 / 16 - Vy * Vy / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 1) / 4
                                                       + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 1) / 4
                                                       + (1.0 - Ux * Ux / 4) * Vh(u, i, j - 1))
                          + (9.0 / 16 - Vy * Vy / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 1) / 4
                                                         + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 1) / 4
                                                         + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 1))
                          + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 3) / 4
                                                                  + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 3) / 4
                                                                  + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 3))
                          + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 3) / 4
                                                                  + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 3) / 4
                                                                  + (-Ux * Ux / 4 + 1) * Vh(u, i, j - 3));
        }
    }
}

__global__ void FN2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){
    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy - 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 1)
                                             + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 1)
                                             + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 1)
                                             + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 1)) / 2
                          + Vy * (Vy + 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 1)
                                               + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 1)
                                               + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 1)
                                               + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 1)) / 2
                          + (1.0 - Vy * Vy) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j)
                                               + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j)
                                               + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j)
                                               + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j));
        }
    }
}

__global__ void FW2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = Vy * (Vy / 2 - 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 2)
                                                 + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 2)
                                                 + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 2)
                                                 + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 2)) / 4
                          + Vy * (Vy / 2 + 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 2)
                                                   + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 2)
                                                   + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 2)
                                                   + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 2)) / 4
                          + (1.0 - Vy * Vy / 4) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j)
                                                   + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j)
                                                   + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j)
                                                   + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j));
        }
    }
}

__global__ void FF2Kernel(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            V(uh, i, j) = (9.0 / 16 - Vy * Vy / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 1)
                                                       + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 1)
                                                       + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 1)
                                                       + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 1))
                          + (-Vy * Vy / 16 + 9.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 1)
                                                          + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 1)
                                                          + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 1)
                                                          + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 1))
                          + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 3)
                                                                  + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 3)
                                                                  + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 3)
                                                                  + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 3))
                          + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 3)
                                                                  +(9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 3)
                                                                  + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 3)
                                                                  + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 3));
        }
    }
}


__global__ void LW2Dkernel1(HaloArray3D* u, HaloArray3D* uh, double Ux, double Vy, unsigned int opt_stset){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            double sts[9];
            int count = 0;
            if (STS_OPT(ST_NN2))
                sts[count++] = Vy * (Vy - 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 1) / 2
                                                  + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 1) / 2
                                                  + (1.0 - Ux * Ux) * Vh(u, i, j + 1)) / 2
                               + Vy * (Vy + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 1) / 2
                                                    + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 1) / 2
                                                    + (1.0 - Ux * Ux) * Vh(u, i, j - 1)) / 2
                               + (1.0 - Vy * Vy) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j) / 2
                                                    + Ux * (Ux + 1.0) * Vh(u, i - 1, j) / 2
                                                    + (1.0 - Ux * Ux) * Vh(u, i, j));

            if (STS_OPT(ST_NF2))
                sts[count++] = (-Vy * Vy / 16 + 9.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 1) / 2
                                                             + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 1) / 2
                                                             + (1.0 - Ux * Ux) * Vh(u, i, j - 1))
                               + (-Vy * Vy / 16 + 9.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 1) / 2
                                                               + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 1) / 2
                                                               + (1.0 - Ux * Ux) * Vh(u, i, j + 1))
                               + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 3) / 2
                                                                       + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 3) / 2
                                                                       + (1.0 - Ux * Ux) * Vh(u, i, j + 3))
                               + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 3) / 2
                                                                       + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 3) / 2
                                                                       + (1.0 - Ux * Ux) * Vh(u, i, j - 3));

            if (STS_OPT(ST_WW2))
                sts[count++] = Vy * (Vy / 2 - 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 2) / 4
                                                      + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 2) / 4
                                                      + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 2)) / 4
                               + Vy * (Vy / 2 + 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 2) / 4
                                                        + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 2) / 4
                                                        + (1.0 - Ux * Ux / 4) * Vh(u, i, j - 2)) / 4
                               + (1.0 - Vy * Vy / 4) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j) / 4
                                                        + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j) / 4
                                                        + (1.0 - Ux * Ux / 4) * Vh(u, i, j));

            if (STS_OPT(ST_FW2))
                sts[count++] = Vy * (Vy / 2 - 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 2)
                                                      + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 2)
                                                      + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 2)
                                                      + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 2)) / 4
                               + Vy * (Vy / 2 + 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 2)
                                                        + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 2)
                                                        + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 2)
                                                        + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 2)) / 4
                               + (1.0 - Vy * Vy / 4) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j)
                                                        + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j)
                                                        + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j)
                                                        + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j));

            if (STS_OPT(ST_WF2))
                sts[count++] = (9.0 / 16 - Vy * Vy / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 1) / 4
                                                            + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 1) / 4
                                                            + (1.0 - Ux * Ux / 4) * Vh(u, i, j - 1))
                               + (9.0 / 16 - Vy * Vy / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 1) / 4
                                                              + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 1) / 4
                                                              + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 1))
                               + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 3) / 4
                                                                       + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 3) / 4
                                                                       + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 3))
                               + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 3) / 4
                                                                       + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 3) / 4
                                                                       + (-Ux * Ux / 4 + 1) * Vh(u, i, j - 3));

            if (STS_OPT(ST_NW2))
                sts[count++] = Vy * (Vy / 2 - 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j + 2) / 2
                                                      + Ux * (Ux + 1.0) * Vh(u, i - 1, j + 2) / 2
                                                      + (1.0 - Ux * Ux) * Vh(u, i, j + 2)) / 4
                               + Vy * (Vy / 2 + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j - 2) / 2
                                                        + Ux * (Ux + 1.0) * Vh(u, i - 1, j - 2) / 2
                                                        + (1.0 - Ux * Ux) * Vh(u, i, j - 2)) / 4
                               + (-Vy * Vy / 4 + 1.0) * (Ux * (Ux - 1.0) * Vh(u, i + 1, j) / 2
                                                         + Ux * (Ux + 1.0) * Vh(u, i - 1, j) / 2
                                                         + (1.0 - Ux * Ux) * Vh(u, i, j));

            if (STS_OPT(ST_WN2))
                sts[count++] = Vy * (Vy - 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j + 1) / 4
                                                  + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j + 1) / 4
                                                  + (1.0 - Ux * Ux / 4) * Vh(u, i, j + 1)) / 2
                               + Vy * (Vy + 1.0) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j - 1) / 4
                                                    + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j - 1) / 4
                                                    + (-Ux * Ux / 4 + 1.0) * Vh(u, i, j - 1)) / 2
                               + (1.0 - Vy * Vy) * (Ux * (Ux / 2 - 1.0) * Vh(u, i + 2, j) / 4
                                                    + Ux * (Ux / 2 + 1.0) * Vh(u, i - 2, j) / 4
                                                    + (1.0 - Ux * Ux / 4) * Vh(u, i, j));

            if (STS_OPT(ST_FN2))
                sts[count++] = Vy * (Vy - 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 1)
                                                  + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 1)
                                                  + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 1)
                                                  + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 1)) / 2
                               + Vy * (Vy + 1.0) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 1)
                                                    + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 1)
                                                    + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 1)
                                                    + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 1)) / 2
                               + (1.0 - Vy * Vy) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j)
                                                    + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j)
                                                    + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j)
                                                    + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j));

            if (STS_OPT(ST_FF2))
                sts[count++] = (9.0 / 16 - Vy * Vy / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 1)
                                                            + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 1)
                                                            + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 1)
                                                            + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 1))
                               + (-Vy * Vy / 16 + 9.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 1)
                                                               + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 1)
                                                               + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 1)
                                                               + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 1))
                               + (Vy * Vy / 16 - Vy / 6 - 1.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j + 3)
                                                                       + (9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j + 3)
                                                                       + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j + 3)
                                                                       + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j + 3))
                               + (Vy * Vy / 16 + Vy / 6 - 1.0 / 16) * ((9.0 / 16 - Ux * Ux / 16) * Vh(u, i - 1, j - 3)
                                                                       +(9.0 / 16 - Ux * Ux / 16) * Vh(u, i + 1, j - 3)
                                                                       + (Ux * Ux / 16 - Ux / 6 - 1.0 / 16) * Vh(u, i + 3, j - 3)
                                                                       + (Ux * Ux / 16 + Ux / 6 - 1.0 / 16) * Vh(u, i - 3, j - 3));

            if (count == 1)
                V(uh, i, j) = sts[count - 1];
            else{
                if ((count % 2) == 0 ) {
                    V(uh, i, j) = (sts[count / 2] + sts[(count + 1) / 2]) / 2;
                }
                else
                    V(uh, i, j) =  sts[count / 2];
            }
        }
    }
}

__global__ void LW2Dkernel2(HaloArray3D* u, HaloArray3D* uh){

    int i0 = blockIdx.x * blockDim.x + threadIdx.x, di = blockDim.x*gridDim.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y, dj = blockDim.y*gridDim.y;
    int x = i0 + j0 * di, total = di * dj;
    for (int j = x; j < u->l.x; j += total) {
        for(int i = 0; i < u->l.y; i++){
            Vh(u, i, j) = Vh(uh, i, j);
        }
    }
}

void Advect3D::updateLW2DCuda(HaloArray3D *u){
    double Ux = V.x * dt / delta.x, Vy = V.y * dt / delta.y;
    const dim3 gridSize(gdim, gdim, 1);
    const dim3 blockSize(bdim, bdim, 1);
    HaloArray3D *uh = new HaloArray3D(Vec3D<int>(u->s.x - 1, u->s.y - 1, 1), Vec3D<int>(0), 1);
    int sizeuh = uh->s.prod();
    HaloArray3D* d_uh;
    double* uuh;
    hipMallocManaged((void**) &d_uh, sizeof(HaloArray3D));
    hipMallocManaged((void**) &uuh, sizeof(double) * sizeuh);
    d_uh->u = uuh;
    d_uh->l = uh->l;
    d_uh->s = uh->s;
    d_uh->halo = uh->halo;
    d_uh->B = uh->B;
    switch (opt_stset){
        case STB_NN2:
            NN2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_NW2:
            NW2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_NF2:
            NF2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_WN2:
            WN2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_WW2:
            WW2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_WF2:
            WF2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_FN2:
            FN2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_FW2:
            FW2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        case STB_FF2:
            FF2Kernel<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy);
            break;
        default:
            LW2Dkernel1<<<gridSize, blockSize>>>(u, d_uh, Ux, Vy, opt_stset);
    }

    hipDeviceSynchronize();
    LW2Dkernel2<<<gridSize, blockSize>>>(u, d_uh);
    hipDeviceSynchronize();
    hipFree(d_uh);
    hipFree(uuh);
}

double Advect3D::simulateAdvectionCuda(HaloArray3D* u, double dtA){
    const dim3 gridSize(gdim, gdim, 1);
    const dim3 blockSize(bdim, bdim, 1);
    double t = 0.0;
    int s = 0;
    HaloArray3D* d_u;
    double* uu;
    int sizeu;
    if(opt_tmr){
        sizeu = u->s.prod() * 3;
        hipMallocManaged((void**) &d_u, sizeof(HaloArray3D));
        hipMallocManaged((void**) &uu, sizeof(double) * sizeu);
        for(int i = 0; i < sizeu; i++)
            uu[i] = u->u[i];
        d_u->u = uu;
        d_u->l = u->l;
        d_u->s = u->s;
        d_u->halo = u->halo;
        d_u->B = u->B;

        while (t < dtA) {
            updateLW2D_tmr_Cuda(d_u);
            boundary<<<gridSize, blockSize>>>(d_u);
            hipDeviceSynchronize();
            t += dt; s++;
        }
    }else{
        sizeu = u->s.prod();
        hipMallocManaged((void**) &d_u, sizeof(HaloArray3D));
        hipMallocManaged((void**) &uu, sizeof(double) * sizeu);
        for(int i = 0; i < sizeu; i++)
            uu[i] = u->u[i];
        d_u->u = uu;
        d_u->l = u->l;
        d_u->s = u->s;
        d_u->halo = u->halo;
        d_u->B = u->B;

        if (opt_stset == STC_XX) {
            while (t < dtA) {
                updateLWN2Cuda(d_u);
                boundary<<<gridSize, blockSize>>>(d_u);
                hipDeviceSynchronize();
                t += dt; s++;
            }
        } else if (opt_stset == STC_XX4) {
            while(t < dtA){
                updateLWN4Cuda(d_u);
                boundary<<<gridSize, blockSize>>>(d_u);
                hipDeviceSynchronize();
                t += dt; s++;
            }
        }else{
            while(t < dtA){
                updateLW2DCuda(d_u);
                boundary<<<gridSize, blockSize>>>(d_u);
                hipDeviceSynchronize();
                t += dt; s++;
            }
        }
    }

    for(int i = 0; i < sizeu; i++) {
        u->u[i] = d_u->u[i];
    }
    hipFree(d_u);
    hipFree(uu);
    return t;
}